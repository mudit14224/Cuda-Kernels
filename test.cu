
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void addKernel(int *c, const int *a, const int *b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main() {
    const int arraySize = 5;
    const int a[arraySize] = {1, 2, 3, 4, 5};
    const int b[arraySize] = {10, 20, 30, 40, 50};
    int c[arraySize] = {0};

    // Device arrays
    int *dev_a = nullptr;
    int *dev_b = nullptr;
    int *dev_c = nullptr;

    // Allocate device memory
    hipMalloc((void **)&dev_a, arraySize * sizeof(int));
    hipMalloc((void **)&dev_b, arraySize * sizeof(int));
    hipMalloc((void **)&dev_c, arraySize * sizeof(int));

    // Copy data from host to device
    hipMemcpy(dev_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, arraySize * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    addKernel<<<1, arraySize>>>(dev_c, dev_a, dev_b);

    // Copy result back to host
    hipMemcpy(c, dev_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // Output result
    for (int i = 0; i < arraySize; ++i) {
        std::cout << c[i] << ' ';
    }
    std::cout << std::endl;

    return 0;
}
